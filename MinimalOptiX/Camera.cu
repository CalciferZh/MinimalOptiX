#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(rtObject, topObject, , );
rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(uint, nSuperSampling, , );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(float, rayEpsilonT, , );

rtDeclareVariable(CamParams, camParams, , );
rtBuffer<uchar4, 2> outputBuffer;

RT_PROGRAM void pinholeCamera() {
  float3 accu = make_float3(0.f, 0.f, 0.f);
  Ray ray;
  ray.origin = camParams.origin;
  ray.ray_type = rayTypeRadiance;
  ray.tmin = rayEpsilonT;
  ray.tmax = RT_DEFAULT_MAX;
  Payload pld;
  pld.depth = 1;
  float2 unit = 1 / make_float2(launchDim);
  float2 xy = make_float2(launchIdx) * unit;
  for (int i = 0; i < nSuperSampling; ++i) {
    pld.randSeed = i * launchDim.x * launchDim.y + \
                   launchIdx.x * launchDim.y + launchIdx.y + 960822;
    pld.color = make_float3(1.f, 1.f, 1.f);
    ray.direction = normalize(
      camParams.srcLowerLeftCorner + \
      (xy.x + (rand(pld.randSeed) - 0.5) * unit.x) * camParams.horizontal + \
      (xy.y + (rand(pld.randSeed) - 0.5) * unit.y) * camParams.vertical - \
      camParams.origin
    );
    rtTrace(topObject, ray, pld);
    accu += pld.color;
  }
  accu /= (float)nSuperSampling;
  outputBuffer[launchIdx] = make_color(accu);
}
