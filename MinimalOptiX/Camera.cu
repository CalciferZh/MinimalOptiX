#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(rtObject, topObject, , );
rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(uint, nSample, , );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(uint, nSuperSampling, , );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(float, rayEpsilonT, , );

rtDeclareVariable(CamParams, camParams, , );
rtBuffer<float3, 2> accuBuffer;
rtBuffer<uchar4, 2> outputBuffer;

RT_PROGRAM void pinholeCamera() {
  Payload pld;
  pld.depth = 1;
  pld.randSeed = nSample * launchDim.x * launchDim.y + launchIdx.x * launchDim.y + launchIdx.x + 960822;
  pld.color = make_float3(1.f);

  Ray ray;
  ray.origin = camParams.origin;
  ray.ray_type = rayTypeRadiance;
  ray.tmin = rayEpsilonT;
  ray.tmax = RT_DEFAULT_MAX;
  float2 xy = (make_float2(launchIdx) + make_float2(rand(pld.randSeed), rand(pld.randSeed)) - 0.5f) / make_float2(launchDim);
  ray.direction = normalize(
    camParams.srcLowerLeftCorner + xy.x * camParams.horizontal + xy.y * camParams.vertical - camParams.origin
  );

  rtTrace(topObject, ray, pld);

  accuBuffer[launchIdx] += pld.color;
  outputBuffer[launchIdx] = make_color(pld.color);
}
