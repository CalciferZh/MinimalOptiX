#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(rtObject, topGroup, , );
rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(int, randSeed, , );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(uint, nSuperSampling, , );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(float, rayEpsilonT, , );

rtDeclareVariable(CamParams, camParams, , );
rtBuffer<float3, 2> accuBuffer;
rtBuffer<uchar4, 2> outputBuffer;

RT_PROGRAM void pinholeCamera() {
  Payload pld;
  pld.depth = 1;
  pld.randSeed = tea<16>(launchIdx.x * launchDim.x + launchIdx.y, randSeed);
  pld.color = make_float3(1.f);

  Ray ray;
  ray.origin = camParams.origin;
  ray.ray_type = rayTypeRadiance;
  ray.tmin = rayEpsilonT;
  ray.tmax = RT_DEFAULT_MAX;
  float2 xy = (make_float2(launchIdx) + make_float2(rand(pld.randSeed), rand(pld.randSeed)) - 0.5f) / make_float2(launchDim);
  ray.direction = normalize(
    camParams.srcLowerLeftCorner + xy.x * camParams.horizontal + xy.y * camParams.vertical - camParams.origin
  );

  rtTrace(topGroup, ray, pld);

  float expo = 1.f / 2.2f;
  pld.color.x = pow(pld.color.x, expo);
  pld.color.y = pow(pld.color.y, expo);
  pld.color.z = pow(pld.color.z, expo);
  pld.color = clamp(pld.color, make_float3(0.f), make_float3(1.f));

  accuBuffer[launchIdx] += pld.color;
  outputBuffer[launchIdx] = make_color(pld.color);
}
