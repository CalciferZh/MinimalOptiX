#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "Structures.h"

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(PayloadRadiance, pldR, rtPayload, );

rtDeclareVariable(uint, rayTypeRadience, , );
rtDeclareVariable(rtObject, topObject, , );
rtDeclareVariable(float, rayEpsilonT, , );
rtDeclareVariable(float3, origin, , );
rtDeclareVariable(float3, horizontal, , );
rtDeclareVariable(float3, vertical, , );
rtDeclareVariable(float3, scrLowerLeftCorner, , );
rtBuffer<uchar4, 2> outputBuffer;

RT_PROGRAM void pinholeCamera() {
  float2 xy = make_float2(launchIdx) / make_float2(launchDim);
  float3 rayOri = origin;
  float3 rayDir = normalize(scrLowerLeftCorner + xy.x * horizontal + xy.y * vertical - origin);
  Ray ray(rayOri, rayDir, rayTypeRadience, rayEpsilonT);
  PayloadRadiance pldR;
  pldR.color = make_float3(1.f);
  pldR.intensity = 1.f;
  rtTrace(topObject, ray, pldR);
  outputBuffer[launchIdx] = make_uchar4(
    static_cast<unsigned char>(__saturatef(pldR.color.z)*255.99f),
    static_cast<unsigned char>(__saturatef(pldR.color.y)*255.99f),
    static_cast<unsigned char>(__saturatef(pldR.color.x)*255.99f),
    255u
  );
}
