#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(rtObject, topGroup, , );
rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(int, randSeed, , );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(uint, nSuperSampling, , );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(float, rayEpsilonT, , );

rtBuffer<float3, 2> accuBuffer;

rtDeclareVariable(CamParams, camParams, , );

RT_PROGRAM void camera() {
  Payload pld;
  pld.depth = 1;
  pld.randSeed = tea<16>(launchIdx.y * launchDim.x + launchIdx.x, randSeed);
  pld.color = make_float3(1.f);

  float3 randInLens = camParams.lensRadius * randInUnitDisk(pld.randSeed);
  float3 offset = camParams.u * randInLens.x + camParams.v * randInLens.y;
  float2 xy = (make_float2(launchIdx) + make_float2(rand(pld.randSeed), rand(pld.randSeed)) - 0.5f) / make_float2(launchDim);
  Ray ray(
    camParams.origin + offset,
    normalize(camParams.scrLowerLeftCorner + xy.x * camParams.horizontal + xy.y * camParams.vertical - camParams.origin - offset),
    rayTypeRadiance,
    rayEpsilonT
  );

  rtTrace(topGroup, ray, pld);

  pld.color = clamp(pld.color, make_float3(0.f), make_float3(1.f));

  accuBuffer[launchIdx] += pld.color;
}
