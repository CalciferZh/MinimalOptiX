#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "Structures.h"

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtDeclareVariable(float, rayEpsilonT, , );
rtDeclareVariable(float3, origin, , );
rtDeclareVariable(float3, u, , );
rtDeclareVariable(float3, v, , );
rtDeclareVariable(float3, w, , );
rtDeclareVariable(float3, scrLowerLeftCorner, , );
rtDeclareVariable(uint, rayTypeRadience, , );
rtDeclareVariable(uint, rayTypeShadow, , );
rtDeclareVariable(PayloadRadiance, pldR, rtPayload, );
rtDeclareVariable(rtObject, topObject, , );
rtBuffer<uchar, 2> outputBuffer;

RT_PROGRAM void pinholeCamera() {
  size_t2 screen = outputBuffer.size();
  float2 xy = make_float2(launchIdx) / make_float2(screen) * 2.f - 1.f;
  float3 rayOri = origin;
  float3 rayDir = normalize(xy.x * u + xy.y * v + w);
  Ray ray(rayOri, rayDir, rayTypeRadience, rayEpsilonT);
  PayloadRadiance pldR;
  pldR.color = make_float3(1.f);
  pldR.intensity = 1.f;
  pldR.depth = 0;
  rtTrace(topObject, ray, pldR);
  outputBuffer[launchIdx] = make_uchar4(
    static_cast<unsigned char>(__saturatef(pldR.color.z)*255.99f),
    static_cast<unsigned char>(__saturatef(pldR.color.y)*255.99f),
    static_cast<unsigned char>(__saturatef(pldR.color.x)*255.99f),
    255u
  );
}
