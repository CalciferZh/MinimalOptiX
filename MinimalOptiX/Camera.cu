#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(rtObject, topGroup, , );
rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(int, randSeed, , );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(uint, nSuperSampling, , );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(float, rayEpsilonT, , );

rtDeclareVariable(CamParams, camParams, , );
rtBuffer<float3, 2> accuBuffer;
rtBuffer<uchar4, 2> outputBuffer;

RT_PROGRAM void pinholeCamera() {
  Payload pld;
  pld.depth = 1;
  pld.randSeed = tea<16>(launchIdx.x * launchDim.x + launchIdx.y, randSeed);
  pld.color = make_float3(1.f);

  Ray ray;
  ray.origin = camParams.origin;
  ray.ray_type = rayTypeRadiance;
  ray.tmin = rayEpsilonT;
  ray.tmax = RT_DEFAULT_MAX;
  float2 xy = (make_float2(launchIdx) + make_float2(rand(pld.randSeed), rand(pld.randSeed)) - 0.5f) / make_float2(launchDim);
  ray.direction = normalize(
    camParams.srcLowerLeftCorner + xy.x * camParams.horizontal + xy.y * camParams.vertical - camParams.origin
  );

  rtTrace(topGroup, ray, pld);

  accuBuffer[launchIdx] += pld.color;
  outputBuffer[launchIdx] = make_color(pld.color);
}
