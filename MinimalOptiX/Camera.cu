#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(rtObject, topObject, , );
rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(float, rayEpsilonT, , );

rtDeclareVariable(CamParams, camParams, , );
rtBuffer<uchar4, 2> outputBuffer;

RT_PROGRAM void pinholeCamera() {
  float2 xy = make_float2(launchIdx) / make_float2(launchDim);
  float3 rayOri = camParams.origin;
  float3 rayDir = normalize(
    camParams.srcLowerLeftCorner + \
    xy.x * camParams.horizontal + \
    xy.y * camParams.vertical - \
    camParams.origin
  );
  Ray ray(rayOri, rayDir, rayTypeRadiance, rayEpsilonT);
  Payload pld;
  pld.color = make_float3(1.f, 1.f, 1.f);
  pld.depth = 1;
  pld.randSeed = launchIdx.x + launchIdx.y * launchDim.x + 960822;
  rtTrace(topObject, ray, pld);
  outputBuffer[launchIdx] = make_color(pld.color);
}
