#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(PayloadRadiance, pldR, rtPayload, );
rtDeclareVariable(uint, rayTypeRadience, , );
rtDeclareVariable(rtObject, topObject, , );
rtDeclareVariable(float, rayEpsilonT, , );
rtDeclareVariable(float3, origin, , );
rtDeclareVariable(float3, horizontal, , );
rtDeclareVariable(float3, vertical, , );
rtDeclareVariable(float3, scrLowerLeftCorner, , );
rtBuffer<uchar4, 2> outputBuffer;

RT_PROGRAM void pinholeCamera() {
  float2 xy = make_float2(launchIdx) / make_float2(launchDim);
  float3 rayOri = origin;
  float3 rayDir = normalize(scrLowerLeftCorner + xy.x * horizontal + xy.y * vertical - origin);
  Ray ray(rayOri, rayDir, rayTypeRadience, rayEpsilonT);
  PayloadRadiance pldR;
  pldR.color = make_float3(1.f, 1.f, 1.f);
  pldR.depth = 1;
  pldR.randSeed = launchIdx.x + launchIdx.y * launchDim.x + 960822;
  rtTrace(topObject, ray, pldR);
  outputBuffer[launchIdx] = make_color(pldR.color);
}
