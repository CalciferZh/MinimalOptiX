#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(float3, badColor, , );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtBuffer<float3, 2> accuBuffer;

RT_PROGRAM void exception() {
  accuBuffer[launchIdx] += badColor;
}
