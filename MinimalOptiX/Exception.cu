#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(float3, badColor, , );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtBuffer<uchar4, 2> outputBuffer;

RT_PROGRAM void exception() {
  const unsigned int code = rtGetExceptionCode();
  rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launchIdx.x, launchIdx.y);
  outputBuffer[launchIdx] = make_color(badColor);
}
