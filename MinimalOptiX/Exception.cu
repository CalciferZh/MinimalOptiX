#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float3, badColor, , );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtBuffer<uchar4, 2> outputBuffer;

RT_PROGRAM void exception() {
  outputBuffer[launchIdx] = make_uchar4(
    static_cast<unsigned char>(__saturatef(badColor.z)*255.99f),
    static_cast<unsigned char>(__saturatef(badColor.y)*255.99f),
    static_cast<unsigned char>(__saturatef(badColor.x)*255.99f),
    255u
  );
}
