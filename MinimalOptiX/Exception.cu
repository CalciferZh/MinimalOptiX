#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "Utils.h"

using namespace optix;

rtDeclareVariable(float3, badColor, , );
rtDeclareVariable(uint2, launchIdx, rtLaunchIndex, );
rtBuffer<uchar4, 2> outputBuffer;

RT_PROGRAM void exception() {
  outputBuffer[launchIdx] = makeColor(badColor);
}
