#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );

// ================= sphere ===================

rtDeclareVariable(float, radius, , );
rtDeclareVariable(float3, center, , );

RT_PROGRAM void sphereIntersect(int) {
  float3 oc = ray.origin - center;
  float a = 1.f;
  float b = dot(ray.direction, oc);
  float c = dot(oc, oc) - radius * radius;
  float discriminant = b * b - c;
  if (discriminant < 0) {
    return;
  }
  float t;
  float squareRoot = sqrt(discriminant);
  t = -b - squareRoot;
  bool checkSecond = true;
  if (rtPotentialIntersection(t)) {
    geoNormal = normalize(ray.origin + t * ray.direction - center);
    if (rtReportIntersection(0)) {
      checkSecond = false;
    }
  }
  if (checkSecond) {
    t = -b + squareRoot;
    if (rtPotentialIntersection(t)) {
      geoNormal = normalize(ray.origin + t * ray.direction - center);
      rtReportIntersection(0);
    }
  }
}

RT_PROGRAM void sphereBBox(int, float result[6]) {
  Aabb* aabb = (Aabb*)result;
  aabb->set(center + radius, center - radius);
}

// ================= quad ======================
// directly copied from nVidia official sample, with code style modification

rtDeclareVariable(float4, plane, , );
rtDeclareVariable(float3, v1, , );
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, anchor, , );

RT_PROGRAM void quadIntersect(int) {
  float3 n = make_float3(plane);
  float dt = dot(ray.direction, n);
  float t = (plane.w - dot(n, ray.origin)) / dt;
  if (t > ray.tmin && t < ray.tmax) {
    float3 p = ray.origin + ray.direction * t;
    float3 vi = p - anchor;
    float a1 = dot(v1, vi);
    if(a1 >= 0 && a1 <= 1){
      float a2 = dot(v2, vi);
      if(a2 >= 0 && a2 <= 1){
        if(rtPotentialIntersection(t)) {
          geoNormal = n;
          rtReportIntersection(0);
        }
      }
    }
  }
}

RT_PROGRAM void quadBBox(int, float result[6]) {
  // v1 and v2 are scaled by 1./length^2.
  // Rescale back to normal for the bounds computation.
  const float3 tv1 = v1 / dot(v1, v1);
  const float3 tv2 = v2 / dot(v2, v2);
  const float3 p00 = anchor;
  const float3 p01 = anchor + tv1;
  const float3 p10 = anchor + tv2;
  const float3 p11 = anchor + tv1 + tv2;
  const float  area = length(cross(tv1, tv2));
  optix::Aabb* aabb = (optix::Aabb*)result;
  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf(fminf(p00, p01), fminf(p10, p11));
    aabb->m_max = fmaxf(fmaxf(p00, p01), fmaxf(p10, p11));
  } else {
    aabb->invalidate();
  }
}
