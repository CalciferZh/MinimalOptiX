#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, radius, , );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );
rtDeclareVariable(float3, center, , );

RT_PROGRAM void sphereIntersect(int) {
  float3 oc = ray.origin - center;
  float a = 1.f;
  float b = dot(ray.direction, oc);
  float c = dot(oc, oc) - radius * radius;
  float discriminant = b * b - c;
  if (discriminant < 0) {
    return;
  }
  float t;
  float squareRoot = sqrt(discriminant);
  t = -b - squareRoot;
  bool checkSecond = true;
  if (rtPotentialIntersection(t)) {
    geoNormal = normalize(ray.origin + t * ray.direction - center);
    if (rtReportIntersection(0)) {
      checkSecond = false;
    }
  }
  if (checkSecond) {
    t = -b + squareRoot;
    if (rtPotentialIntersection(t)) {
      geoNormal = normalize(ray.origin + t * ray.direction - center);
      rtReportIntersection(0);
    }
  }
}

RT_PROGRAM void sphereBBox(int, float result[6]) {
  Aabb* aabb = (Aabb*)result;
  aabb->set(center + radius, center - radius);
}

