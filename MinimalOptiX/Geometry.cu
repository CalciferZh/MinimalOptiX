#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(Ray,    ray,       rtCurrentRay,        );
rtDeclareVariable(float,  radius,    ,                    );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );
rtDeclareVariable(float3, center,    ,                    );

RT_PROGRAM void sphereIntersect(int) {
  float3 oc = ray.origin - center;
  float a = dot(ray.direction, ray.direction);
  float b = dot(oc, ray.direction);
  float c = dot(oc, oc) - radius * radius;
  float discriminant = b * b - a * c;
  float t;
  if (discriminant < 0) {
    return;
  }
  float squareRoot = sqrt(discriminant);
  t = (-b - squareRoot) / a;
  bool checkSecond = true;
  if (rtPotentialIntersection(t)) {
    if (rtReportIntersection(0)) {
      checkSecond = false;
    }
  }
  if (checkSecond) {
    t = (-b + squareRoot) / a;
    if (rtPotentialIntersection(t)) {
      rtReportIntersection(0);
    }
  }
}

RT_PROGRAM void sphereBBox(int, float result[6]) {
  Aabb* aabb = (Aabb*)result;
  aabb->set(radius + center, radius - center);
}

