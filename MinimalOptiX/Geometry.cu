#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );
rtDeclareVariable(float3, shadingNormal,   attribute shadingNormal, );

// ==================== sphere ===================

rtDeclareVariable(SphereParams, sphereParams, , );

RT_PROGRAM void sphereIntersect(int) {
  float3 oc = ray.origin - sphereParams.center;
  float b = dot(ray.direction, oc);
  float c = dot(oc, oc) - sphereParams.radius * sphereParams.radius;
  float discriminant = b * b - c;
  if (discriminant < 0) {
    return;
  }
  float t;
  float squareRoot = sqrt(discriminant);
  t = -b - squareRoot;
  bool checkSecond = true;
  if (rtPotentialIntersection(t)) {
    geoNormal = normalize(
      ray.origin + t * ray.direction - sphereParams.center
    );
    if (rtReportIntersection(0)) {
      checkSecond = false;
    }
  }
  if (checkSecond) {
    t = -b + squareRoot;
    if (rtPotentialIntersection(t)) {
      geoNormal = normalize(
        ray.origin + t * ray.direction - sphereParams.center
      );
      rtReportIntersection(0);
    }
  }
}

RT_PROGRAM void sphereBBox(int, float result[6]) {
  Aabb* aabb = (Aabb*)result;
  aabb->set(
    sphereParams.center + sphereParams.radius,
    sphereParams.center - sphereParams.radius
  );
}

// ==================== quad ======================
// directly copied from nVidia official sample, with code style modification

rtDeclareVariable(QuadParams, quadParams, , );

RT_PROGRAM void quadIntersect(int) {
  float3 n = make_float3(quadParams.plane);
  float dt = dot(ray.direction, n);
  float t = (quadParams.plane.w - dot(n, ray.origin)) / dt;
  if (t > ray.tmin && t < ray.tmax) {
    float3 p = ray.origin + ray.direction * t;
    float3 vi = p - quadParams.anchor;
    float a1 = dot(quadParams.v1, vi);
    if(a1 >= 0 && a1 <= 1){
      float a2 = dot(quadParams.v2, vi);
      if(a2 >= 0 && a2 <= 1){
        if(rtPotentialIntersection(t)) {
          geoNormal = n;
          rtReportIntersection(0);
        }
      }
    }
  }
}

RT_PROGRAM void quadBBox(int, float result[6]) {
  // v1 and v2 are scaled by 1./length^2.
  // Rescale back to normal for the bounds computation.
  const float3 tv1 = quadParams.v1 / dot(quadParams.v1, quadParams.v1);
  const float3 tv2 = quadParams.v2 / dot(quadParams.v2, quadParams.v2);
  const float3 p00 = quadParams.anchor;
  const float3 p01 = quadParams.anchor + tv1;
  const float3 p10 = quadParams.anchor + tv2;
  const float3 p11 = quadParams.anchor + tv1 + tv2;
  const float  area = length(cross(tv1, tv2));
  optix::Aabb* aabb = (optix::Aabb*)result;
  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf(fminf(p00, p01), fminf(p10, p11));
    aabb->m_max = fmaxf(fmaxf(p00, p01), fmaxf(p10, p11));
  } else {
    aabb->invalidate();
  }
}

// ==================== mesh ======================

rtBuffer<float3> vertexBuffer;
rtBuffer<float3> normalBuffer;
rtBuffer<float2> texcoordBuffer;
rtBuffer<int3>   indexBuffer;

rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, backHitPoint, attribute backHitPoint, );
rtDeclareVariable(float3, frontHitPoint, attribute frontHitPoint, );
rtDeclareVariable(float, t, rtIntersectionDistance, );

RT_PROGRAM void meshIntersect(int primIdx) {
  const int3 v_idx = indexBuffer[primIdx];
  const float3 p0 = vertexBuffer[v_idx.x];
  const float3 p1 = vertexBuffer[v_idx.y];
  const float3 p2 = vertexBuffer[v_idx.z];

  float3 n;
  float t;
  float beta;
  float gamma;
  if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma)) {
    if (rtPotentialIntersection(t)) {
      geoNormal = normalize(n);
      if(normalBuffer.size() == 0) {
        shadingNormal = geoNormal;
      } else {
        shadingNormal = normalize(normalBuffer[v_idx.y] * beta + normalBuffer[v_idx.z] * gamma + normalBuffer[v_idx.x] * (1.f - beta - gamma));
      }
      if (texcoordBuffer.size() == 0) {
        texcoord = make_float3( 0.0f, 0.0f, 0.0f );
      } else {
        float2 t0 = texcoordBuffer[v_idx.x];
        float2 t1 = texcoordBuffer[v_idx.y];
        float2 t2 = texcoordBuffer[v_idx.z];
        texcoord = make_float3( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );
      }
      refineHitpoint(
        ray.origin + t * ray.direction,
        ray.direction,
        geoNormal,
        p0,
        backHitPoint,
        frontHitPoint
      );
      rtReportIntersection(0);
    }
  }
}

RT_PROGRAM void meshBBox (int primIdx, float result[6]) {
  const int3 v_idx = indexBuffer[primIdx];
  const float3 v0 = vertexBuffer[v_idx.x];
  const float3 v1 = vertexBuffer[v_idx.y];
  const float3 v2 = vertexBuffer[v_idx.z];
  const float area = length(cross(v1 - v0, v2 - v0));
  optix::Aabb* aabb = (optix::Aabb*)result;
  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf(fminf(v0, v1), v2);
    aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
  } else {
    aabb->invalidate();
  }
}
