#include "hip/hip_runtime.h"
#include "Structures.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float3, bgColor, , );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Payload, pld, rtPayload, );

RT_PROGRAM void staticMiss() {
  pld.color *= bgColor;
}
