#include "hip/hip_runtime.h"
#include "Structures.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float3, bgColor, , );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PayloadRadiance, pldR, rtPayload, );

RT_PROGRAM void staticMiss() {
  pldR.color *= bgColor;
}

rtDeclareVariable(float3, gradColorMax, , );
rtDeclareVariable(float3, gradColorMin, , );

RT_PROGRAM void vGradMiss() {
  float r = ray.direction.y * 0.5f + 0.5f;
  pldR.color *= r * gradColorMax + (1 - r) * gradColorMin;
}
