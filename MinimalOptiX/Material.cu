#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "Structures.h"

using namespace optix;

rtDeclareVariable(PayloadRadiance, pldR, rtPayload, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );
rtDeclareVariable(float3, mtlColor, , );
rtBuffer<Light> lights;

RT_PROGRAM void phong() {
  float3 tmpColor = { 0.f, 0.f, 0.f };
  for (int i = 0; i < lights.size(); ++i) {
    float3 P = ray.origin + t * ray.direction;
    float3 PtoL = normalize(lights[i].position - P);
    float brightness = dot(geoNormal, PtoL);
    if (brightness > 0) {
      tmpColor += brightness * lights[i].color;
    }
  }
  pldR.color *= (tmpColor / lights.size());
  pldR.color *= mtlColor;
}
