#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "Structures.h"

using namespace optix;

rtDeclareVariable(PayloadRadiance, pldR, rtPayload, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );
rtDeclareVariable(float3, mtlColor, , );
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, envLightColor, , );
rtBuffer<Light> lights;

RT_PROGRAM void phong() {
  float3 tmpColor = Ka * envLightColor;
  for (int i = 0; i < lights.size(); ++i) {
    float3 P = ray.origin + t * ray.direction;
    float3 PtoL = normalize(lights[i].position - P);
    float brightness = dot(geoNormal, PtoL);
    if (brightness > 0) {
      tmpColor += brightness * lights[i].color;
    }
  }
  pldR.color *= tmpColor;
  pldR.color *= mtlColor;
}
