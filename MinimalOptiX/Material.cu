#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "Structures.h"

using namespace optix;

rtDeclareVariable(PayloadRadiance, pldR, rtPayload, );
rtDeclareVariable(float3, mtlColor, , );

RT_PROGRAM void closestHitStatic() {
  pldR.color *= mtlColor;
}
