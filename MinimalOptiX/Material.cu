#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "disney.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtDeclareVariable(rtObject, topGroup, , );
rtDeclareVariable(uint, rayMaxDepth, , );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(uint, rayTypeShadow, , );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(float, rayEpsilonT, , );
rtDeclareVariable(float, rayMinIntensity, , );
rtDeclareVariable(float3, absorbColor, , );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float3, frontHitPoint, attribute frontHitPoint, );
rtDeclareVariable(float3, backHitPoint, attribute backHitPoint, );

// =================== lambertian ======================

rtDeclareVariable(LambertianParams, lambParams, , );

RT_PROGRAM void lambertian() {
  if (payload.depth > rayMaxDepth || length(payload.color) < rayMinIntensity) {
    payload.color = absorbColor;
    return;
  }
  float3 tmpColor = { 0.f, 0.f, 0.f };
  Ray newRay(
    ray.origin + t * ray.direction,
    normalize(geoNormal + randInUnitSphere(payload.randSeed)),
    rayTypeRadiance,
    rayEpsilonT
  );
  Payload newPayload = folkPayload(payload);
  rtTrace(topGroup, newRay, newPayload);
  payload.color = newPayload.color * lambParams.albedo;
}

// ====================== metal ==========================

rtDeclareVariable(MetalParams, metalParams, , );

RT_PROGRAM void metal() {
  if (payload.depth > rayMaxDepth || length(payload.color) < rayMinIntensity) {
    payload.color = absorbColor;
    return;
  }
  Ray newRay(
    ray.origin + t * ray.direction,
    normalize(reflect(ray.direction, geoNormal) + metalParams.fuzz * randInUnitSphere(payload.randSeed)),
    rayTypeRadiance,
    rayEpsilonT
  );
  Payload newPayload;
  newPayload.depth = payload.depth + 1;
  newPayload.color = make_float3(1.f);
  newPayload.randSeed = tea<16>(payload.randSeed, newPayload.depth);
  rtTrace(topGroup, newRay, newPayload);
  payload.color = metalParams.albedo * newPayload.color;
}

// ====================== glass ==========================

rtDeclareVariable(GlassParams, glassParams, , );

RT_PROGRAM void glass() {
  if (payload.depth > rayMaxDepth || length(payload.color) < rayMinIntensity) {
    payload.color = absorbColor;
    return;
  }

  float3 normal = shadingNormal;
	float cosThetaI = -dot(ray.direction, normal);
	float refIdx;
	if (cosThetaI > 0.f) {
		refIdx = glassParams.refIdx;
	} else {
		refIdx = 1.f / glassParams.refIdx;
		cosThetaI = -cosThetaI;
		normal = -normal;
	}

	float3 refracted;
  float totalReflection = !refract(refracted, ray.direction, normal, refIdx);
	float cosThetaT = -dot(normal, refracted);
	float reflectProb =  totalReflection ? 1.f : fresnel(cosThetaI, cosThetaT, refIdx);
  Ray newRay;
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPayload;
  newPayload.depth = payload.depth + 1;
  newPayload.color = make_float3(1.f);
  newPayload.randSeed = tea<16>(payload.randSeed, newPayload.depth);
  if (rand(payload.randSeed) < reflectProb) {
    newRay.origin = frontHitPoint;
    newRay.direction = reflect(ray.direction, normal);
  } else {
    newRay.origin = backHitPoint;
    newRay.direction = refracted;
  }
  rtTrace(topGroup, newRay, newPayload);
  payload.color = newPayload.color * glassParams.albedo;
}

// ====================== Disney =========================

rtDeclareVariable(DisneyParams, disneyParams, , );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtBuffer<LightParams> lights;

RT_PROGRAM void disney() {
  if (payload.depth > rayMaxDepth || length(payload.color) < rayMinIntensity) {
    payload.color = absorbColor;
    return;
  }

  float3 N, L, V, H;
  N = faceforward(shadingNormal, -ray.direction, geoNormal);
  V = -ray.direction;
  float3 baseColor;
  if (disneyParams.albedoID == RT_TEXTURE_ID_NULL) {
    baseColor = disneyParams.color;
  } else {
    baseColor = make_float3(optix::rtTex2D<float4>(disneyParams.albedoID, texcoord.x, texcoord.y));
  }

  if (disneyParams.brdfType == GLASS) {
    float3 normal = shadingNormal;
    float cosThetaI = -dot(ray.direction, normal);
    float refIdx;
    if (cosThetaI > 0.f) {
      refIdx = 1.45f;
    } else {
      refIdx = 1.f / 1.45f;
      cosThetaI = -cosThetaI;
      normal = -normal;
    }

    float3 refracted;
    float totalReflection = !refract(refracted, ray.direction, normal, refIdx);
    float cosThetaT = -dot(normal, refracted);
    float reflectProb =  totalReflection ? 1.f : fresnel(cosThetaI, cosThetaT, refIdx);
    Ray newRay;
    newRay.ray_type = rayTypeRadiance;
    newRay.tmin = rayEpsilonT;
    newRay.tmax = RT_DEFAULT_MAX;
    Payload newPayload;
    newPayload.depth = payload.depth + 1;
    newPayload.color = make_float3(1.f);
    newPayload.randSeed = tea<16>(payload.randSeed, newPayload.depth);
    if (rand(payload.randSeed) < reflectProb) {
      newRay.origin = frontHitPoint;
      newRay.direction = reflect(ray.direction, normal);
    } else {
      newRay.origin = backHitPoint;
      newRay.direction = refracted;
    }
    rtTrace(topGroup, newRay, newPayload);
    payload.color = newPayload.color * baseColor;
    return;
  }

  // direct light sample
  float3 directLightColor = make_float3(0.f);
  for (int i = 0; i < lights.size(); ++i) {
    LightParams light = lights[i];
    float3 pointOnLight;
    float3 normalOnLight;
    if (light.shape == SPHERE) {
      pointOnLight = light.position + randInUnitSphere(payload.randSeed) * light.radius;
      normalOnLight = normalize(pointOnLight - light.position);
    } else {
      pointOnLight = light.position + light.u * rand(payload.randSeed) + light.v * rand(payload.randSeed);
      normalOnLight = normalize(light.normal);
    }
    L = pointOnLight - frontHitPoint;
    float lightDst = length(L);
    L = normalize(L);
    if (dot(L, N) > 0.f && dot(L, normalOnLight) < 0.f) {
      Ray newRay(frontHitPoint, L, rayTypeShadow, rayEpsilonT, lightDst - rayEpsilonT);
      Payload newPayload;
      newPayload.depth = payload.depth + 1;
      newPayload.attenuation = make_float3(1.f);
      newPayload.randSeed = tea<16>(payload.randSeed, newPayload.depth);
      rtTrace(topGroup, newRay, newPayload);
      if (length(newPayload.attenuation)) {
        H = normalize(L + V);
        float lightPdf = lightDst * lightDst / light.area / dot(normalOnLight, -L);
        float objPdf = disneyPdf(disneyParams, N, L, V, H);
        if (lightPdf > 0 && objPdf > 0) {
          float3 brdf = disneyEval(disneyParams, baseColor, N, L, V, H);
          directLightColor += powerHeuristic(lightPdf, objPdf) * brdf * light.emission * newPayload.attenuation / max(0.001f, lightPdf);
        }
      }
    }
  }

  float3 indirectColor = make_float3(0.f);
  disneySample(payload.randSeed, disneyParams, N, L, V, H);
  if (dot(N, L) > 0.0f && dot(N, V) > 0.0f) {
    Ray newRay(frontHitPoint, L, rayTypeRadiance, rayEpsilonT);
    Payload newPayload;
    newPayload.depth = payload.depth + 1;
    newPayload.color = make_float3(1.f);
    newPayload.randSeed = tea<16>(payload.randSeed, newPayload.depth);
    rtTrace(topGroup, newRay, newPayload);

    float pdf = disneyPdf(disneyParams, N, L, V, H);
    if (pdf > 0) {
      float3 brdf = disneyEval(disneyParams, baseColor, N, L, V, H);
      indirectColor = brdf * newPayload.color / pdf;
    }
  }

  payload.color = indirectColor + directLightColor + disneyParams.emission;
}

RT_PROGRAM void disneyAnyHit() {
  if (disneyParams.brdfType == GLASS) {
    payload.attenuation *= disneyParams.color;
  } else {
    payload.attenuation = make_float3(0.f);
    rtTerminateRay();
  }
}

// ====================== light ==========================

rtDeclareVariable(LightParams, lightParams, , );

RT_PROGRAM void light() {
  payload.color = lightParams.emission;
}

