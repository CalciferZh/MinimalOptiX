#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtDeclareVariable(rtObject, topGroup, , );
rtDeclareVariable(uint, rayMaxDepth, , );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(float, rayEpsilonT, , );
rtDeclareVariable(float, rayMinIntensity, , );
rtDeclareVariable(float3, absorbColor, , );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float3, frontHitPoint, attribute frontHitPoint, );
rtDeclareVariable(float3, backHitPoint, attribute backHitPoint, );

// =================== lambertian ======================

rtDeclareVariable(LambertianParams, lambParams, , );

RT_PROGRAM void lambertian() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  int nNewRay = lambParams.nScatter;
  if (pld.depth > lambParams.scatterMaxDepth) {
    nNewRay = 1;
  }
  float3 tmpColor = { 0.f, 0.f, 0.f };
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.depth = pld.depth + 1;
  for (int i = 0; i < nNewRay; ++i) {
    newRay.direction = normalize(geoNormal + randInUnitSphere(pld.randSeed));
    newPld.color = make_float3(1.f);
    newPld.randSeed = tea<16>(pld.randSeed, newPld.depth * lambParams.nScatter + i);
    rtTrace(topGroup, newRay, newPld);
    tmpColor += newPld.color;
  }
  tmpColor /= nNewRay;
  pld.color = tmpColor * lambParams.albedo;
}

// ====================== metal ==========================

rtDeclareVariable(MetalParams, metalParams, , );

RT_PROGRAM void metal() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.direction = normalize(reflect(ray.direction, geoNormal) + metalParams.fuzz * randInUnitSphere(pld.randSeed));
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.color = make_float3(1.f);
  newPld.depth = pld.depth + 1;
  newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
  rtTrace(topGroup, newRay, newPld);
  pld.color = metalParams.albedo * newPld.color;
}

// ====================== glass ==========================

rtDeclareVariable(GlassParams, glassParams, , );

RT_PROGRAM void glass() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }

  float3 normal = shadingNormal;
	float cosThetaI = -dot(ray.direction, normal);
	float refIdx;
	if (cosThetaI > 0.f) {
		refIdx = glassParams.refIdx;
	} else {
		refIdx = 1.f / glassParams.refIdx;
		cosThetaI = -cosThetaI;
		normal = -normal;
	}

	float3 refracted;
  float totalReflection = !refract(refracted, ray.direction, normal, refIdx);
	float cosThetaT = -dot(normal, refracted);
	float reflectProb =  totalReflection ? 1.f : fresnel(cosThetaI, cosThetaT, refIdx);
  Ray newRay;
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.depth = pld.depth + 1;
  newPld.color = make_float3(1.f, 1.f, 1.f);
  newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
  if (rand(pld.randSeed) < reflectProb) {
    newRay.origin = frontHitPoint;
    newRay.direction = reflect(ray.direction, normal);
  } else {
    newRay.origin = backHitPoint;
    newRay.direction = refracted;
  }
  rtTrace(topGroup, newRay, newPld);
  pld.color = newPld.color * glassParams.albedo;
}

// ====================== Disney =========================

rtDeclareVariable(DisneyParams, disneyParams, , );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

RT_PROGRAM void disney() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }

  float3 baseColor;
  if (disneyParams.albedoID == RT_TEXTURE_ID_NULL) {
    baseColor = disneyParams.color;
  } else {
		baseColor = make_float3(optix::rtTex2D<float4>(disneyParams.albedoID, texcoord.x, texcoord.y));
  }
  // sample
  float3 N = faceforward(shadingNormal, -ray.direction, geoNormal);
  float3 V = -ray.direction;
  float3 L;
  float diffuseRatio = 0.5f * (1.0f - disneyParams.metallic);
  float r1 = rand(pld.randSeed);
  float r2 = rand(pld.randSeed);
  optix::Onb onb(N);
  if (rand(pld.randSeed) < diffuseRatio) { // diffuse
    cosine_sample_hemisphere(r1, r2, L);
    onb.inverse_transform(L);
  } else { // spect
    float a = max(0.001f, disneyParams.roughness);
    float phi = r1 * 2.0f * M_PIf;
    float cosTheta = sqrtf((1.f - r2) / (1.0f + (a * a - 1.f) * r2));
    float sinTheta = sqrtf(1.0f - (cosTheta * cosTheta));
    float sinPhi = sinf(phi);
    float cosPhi = cosf(phi);
    float3 half = make_float3(sinTheta*cosPhi, sinTheta*sinPhi, cosTheta);
    onb.inverse_transform(half);
    L = 2.0f * dot(V, half) * half - V;
  }
  Ray newRay(frontHitPoint, normalize(L), rayTypeRadiance, rayEpsilonT);
  Payload newPld;
  newPld.depth = pld.depth + 1;
  newPld.color = make_float3(1.f);
  newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
  rtTrace(topGroup, newRay, newPld);
  float3 lightColor = newPld.color;

  // probability for this light
  float specularAlpha = max(0.001f, disneyParams.roughness);
  float clearcoatAlpha = lerp(0.1f, 0.001f, disneyParams.clearcoatGloss);
  float specularRatio = 1.f - diffuseRatio;
  float3 half = normalize(L + V);
  float cosTheta = abs(dot(half, N));
  float pdfGTR2 = GTR2(cosTheta, specularAlpha) * cosTheta;
  float pdfGTR1 = GTR1(cosTheta, clearcoatAlpha) * cosTheta;
  // calculate diffuse and specular pdfs and mix ratio
  float ratio = 1.0f / (1.0f + disneyParams.clearcoat);
  float pdfSpec = lerp(pdfGTR1, pdfGTR2, ratio) / (4.0 * abs(dot(L, half)));
  float pdfDiff = abs(dot(L, N))* (1.0f / M_PIf);
  float pdf = diffuseRatio * pdfDiff + specularRatio * pdfSpec;

  if (pdf < 0) {
    pld.color = make_float3(0.f);
    return;
  }

  // evaluate color
  float NDotL = dot(N, L);
  float NDotV = dot(N, V);
  if (NDotL <= 0.0f || NDotV <= 0.0f) {
    pld.color = make_float3(0.f);
    return;
  }
  float3 H = normalize(L + V);
  float NDotH = dot(N, H);
  float LDotH = dot(L, H);
  float luminance = dot(baseColor, make_float3(0.3, 0.6, 0.1));
  float3 Ctint = luminance > 0.f ? baseColor / luminance : make_float3(1.f);
  float3 Cspec0 = lerp(disneyParams.specular * 0.08f * lerp(make_float3(1.f), Ctint, disneyParams.specularTint), baseColor, disneyParams.metallic);
  float3 Csheen = lerp(make_float3(1.f), Ctint, disneyParams.sheenTint);
  float FL = schlickFresnel(NDotL);
  float FV = schlickFresnel(NDotV);
  float Fd90 = 0.5f + 2.f * LDotH * LDotH * disneyParams.roughness;
  float Fd = lerp(1.f, Fd90, FL) * lerp(1.f, Fd90, FV);
  float Fss90 = LDotH * LDotH * disneyParams.roughness;
  float Fss = lerp(1.0f, Fss90, FL) * lerp(1.0f, Fss90, FV);
  float ss = 1.25f * (Fss * (1.f / (NDotL + NDotV) - 0.5f) + 0.5f);
  float aspect = sqrt(1 - disneyParams.anisotropic * 0.9f);
  float ax = max(.001f, square(disneyParams.roughness) / aspect);
  float ay = max(.001f, square(disneyParams.roughness) * aspect);
  float3 X = normalize(onb.m_tangent);
  float3 Y = normalize(cross(shadingNormal, X));
  float Ds = GTR2Aniso(NDotH, dot(H, X), dot(H, Y), ax, ay);
  float FH = schlickFresnel(LDotH);
  float3 Fs = lerp(Cspec0, make_float3(1.f), FH);
  float roughg = square(disneyParams.roughness * 0.5f + 0.5f);
  float Gs  = smithGGgxAniso(NDotL, dot(L, X), dot(L, Y), ax, ay) *
              smithGGgxAniso(NDotV, dot(V, X), dot(V, Y), ax, ay);
  float3 Fsheen = FH * disneyParams.sheen * Csheen;
  float Dr = GTR1(NDotH, lerp(0.1f, 0.001f, disneyParams.clearcoatGloss));
  float Fr = lerp(0.04f, 1.f, FH);
  float Gr = smithGGgx(NDotL, 0.25f) * smithGGgx(NDotV, 0.25f);
  float3 bsdf = ((1.0f / M_PIf) * lerp(Fd, ss, disneyParams.subsurface) * baseColor + Fsheen) * (1.0f - disneyParams.metallic) +
                Gs * Fs * Ds + 0.25f * disneyParams.clearcoat * Gr * Fr * Dr;

  pld.color = bsdf * lightColor / pdf;
  float exp = 1.f / 2.2f;
  pld.color.x = pow(pld.color.x, exp);
  pld.color.y = pow(pld.color.y, exp);
  pld.color.z = pow(pld.color.z, exp);
}

// ====================== light ==========================

rtDeclareVariable(LightParams, lightParams, , );
// NOTE: some of light's attributes need to be computed mannually

RT_PROGRAM void light() {
  pld.color = lightParams.emission * clamp(dot(ray.direction, shadingNormal), 0.f, 1.f);
}
