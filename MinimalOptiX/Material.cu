#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtDeclareVariable(rtObject, topObject, , );
rtDeclareVariable(uint, rayMaxDepth, , );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(float, rayEpsilonT, , );
rtDeclareVariable(float, rayMinIntensity, , );
rtDeclareVariable(float3, absorbColor, , );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );

// =================== lambertian ======================

rtDeclareVariable(LambertianParams, lambParams, , );

RT_PROGRAM void lambertian() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  int nNewRay = lambParams.nScatter;
  if (pld.depth > lambParams.scatterMaxDepth) {
    nNewRay = 1;
  }
  float3 tmpColor = { 0.f, 0.f, 0.f };
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.depth = pld.depth + 1;
  for (int i = 0; i < nNewRay; ++i) {
    newRay.direction = normalize(geoNormal + randInUnitSphere(pld.randSeed));
    newPld.color = make_float3(1.f, 1.f, 1.f);
    newPld.randSeed = tea<16>(pld.randSeed, newPld.depth * lambParams.nScatter + i);
    rtTrace(topObject, newRay, newPld);
    tmpColor += newPld.color;
  }
  tmpColor /= nNewRay;
  pld.color *= tmpColor;
  pld.color *= lambParams.albedo;
}

// ====================== metal ==========================

rtDeclareVariable(MetalParams, metalParams, , );

RT_PROGRAM void metal() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.direction = normalize(reflect(ray.direction, geoNormal) + metalParams.fuzz * randInUnitSphere(pld.randSeed));
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.color = make_float3(1.f, 1.f, 1.f);
  newPld.depth = pld.depth + 1;
  newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
  rtTrace(topObject, newRay, newPld);
  pld.color *= newPld.color;
  pld.color *= metalParams.albedo;
}

// ====================== glass ==========================

rtDeclareVariable(GlassParams, glassParams, , );

RT_PROGRAM void glass() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }

  float3 reflected = reflect(ray.direction, geoNormal);
  float3 outwardNormal;
  float realRefIdx;
  float cosine;
  if (dot(ray.direction, geoNormal) > 0) {
    outwardNormal = -geoNormal;
    realRefIdx = glassParams.refIdx;
    cosine = dot(ray.direction, geoNormal);
    cosine = sqrt(1 - glassParams.refIdx * glassParams.refIdx * (1 - cosine * cosine));
  } else {
    outwardNormal = geoNormal;
    realRefIdx = 1.f / glassParams.refIdx;
    cosine = dot(-ray.direction, geoNormal);
  }
  float3 refracted;
  float reflectProb;
  int nNewRay;
  if (refract(ray.direction, outwardNormal, realRefIdx, refracted)) {
    reflectProb = schlick(cosine, glassParams.refIdx);
    nNewRay = glassParams.nScatter;
  } else {
    reflectProb = 1.f;
    nNewRay = 1;
  }
  if (pld.depth > glassParams.scatterMaxDepth) {
    nNewRay = 1;
  }
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.depth = pld.depth + 1;
  float3 tmpColor = { 0.f, 0.f, 0.f };
  for (int i = 0; i < nNewRay; ++i) {
    if (rand(pld.randSeed) < reflectProb) {
      newRay.direction = reflected;
    } else {
      newRay.direction = refracted;
    }
    newPld.color = make_float3(1.f, 1.f, 1.f);
    newPld.randSeed = tea<16>(pld.randSeed, newPld.depth * glassParams.nScatter + i);
    rtTrace(topObject, newRay, newPld);
    tmpColor += newPld.color;
  }
  tmpColor /= nNewRay;
  pld.color *= tmpColor;
  pld.color *= glassParams.albedo;
}

// ====================== Disney =========================

rtDeclareVariable(int, MaterialID, , );
rtBuffer<DisneyParams> DisneyParamsMtlBuf;


// ====================== light ==========================

rtDeclareVariable(float3, lightColor, , );

RT_PROGRAM void light() {
  pld.color *= lightColor;
}
