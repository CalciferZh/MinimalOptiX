#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "Structures.h"

using namespace optix;

rtDeclareVariable(PayloadRadiance, pldR, rtPayload, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );
rtDeclareVariable(float3, mtlColor, , );
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, phongExp, , );
rtDeclareVariable(float3, envLightColor, , );
rtBuffer<Light> lights;

RT_PROGRAM void phong() {
  float3 tmpColor = Ka * envLightColor;
  for (int i = 0; i < lights.size(); ++i) {
    float3 P = ray.origin + t * ray.direction;
    float3 L = normalize(lights[i].position - P);
    float nDl = dot(geoNormal, L);
    if (nDl > 0) {
      tmpColor += Kd * nDl * lights[i].color;
      float3 H = normalize(L - ray.direction);
      float nDh = dot(geoNormal, H);
      if (nDh > 0) {
        tmpColor += Ks * lights[i].color * pow(nDh, phongExp);
      }
    }
  }
  pldR.color *= tmpColor;
  pldR.color *= mtlColor;
}
