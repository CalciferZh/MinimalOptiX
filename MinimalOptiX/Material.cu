#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtDeclareVariable(rtObject, topGroup, , );
rtDeclareVariable(uint, rayMaxDepth, , );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(float, rayEpsilonT, , );
rtDeclareVariable(float, rayMinIntensity, , );
rtDeclareVariable(float3, absorbColor, , );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float3, frontHitPoint, attribute frontHitPoint, );
rtDeclareVariable(float3, backHitPoint, attribute backHitPoint, );

// =================== lambertian ======================

rtDeclareVariable(LambertianParams, lambParams, , );

RT_PROGRAM void lambertian() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  int nNewRay = lambParams.nScatter;
  if (pld.depth > lambParams.scatterMaxDepth) {
    nNewRay = 1;
  }
  float3 tmpColor = { 0.f, 0.f, 0.f };
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.depth = pld.depth + 1;
  for (int i = 0; i < nNewRay; ++i) {
    newRay.direction = normalize(geoNormal + randInUnitSphere(pld.randSeed));
    newPld.color = make_float3(1.f);
    newPld.randSeed = tea<16>(pld.randSeed, newPld.depth * lambParams.nScatter + i);
    rtTrace(topGroup, newRay, newPld);
    tmpColor += newPld.color;
  }
  tmpColor /= nNewRay;
  pld.color = tmpColor * lambParams.albedo;
}

// ====================== metal ==========================

rtDeclareVariable(MetalParams, metalParams, , );

RT_PROGRAM void metal() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.direction = normalize(reflect(ray.direction, geoNormal) + metalParams.fuzz * randInUnitSphere(pld.randSeed));
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.color = make_float3(1.f);
  newPld.depth = pld.depth + 1;
  newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
  rtTrace(topGroup, newRay, newPld);
  pld.color = metalParams.albedo * newPld.color;
}

// ====================== glass ==========================

rtDeclareVariable(GlassParams, glassParams, , );

RT_PROGRAM void glass() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }

  float3 normal = shadingNormal;
	float cosThetaI = -dot(ray.direction, normal);
	float refIdx;
	if (cosThetaI > 0.f) {
		refIdx = glassParams.refIdx;
	} else {
		refIdx = 1.f / glassParams.refIdx;
		cosThetaI = -cosThetaI;
		normal = -normal;
	}

	float3 refracted;
  float totalReflection = !refract(refracted, ray.direction, normal, refIdx);
	float cosThetaT = -dot(normal, refracted);
	float reflectProb =  totalReflection ? 1.f : fresnel(cosThetaI, cosThetaT, refIdx);
  Ray newRay;
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.depth = pld.depth + 1;
  newPld.color = make_float3(1.f, 1.f, 1.f);
  newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
  if (rand(pld.randSeed) < reflectProb) {
    newRay.origin = frontHitPoint;
    newRay.direction = reflect(ray.direction, normal);
  } else {
    newRay.origin = backHitPoint;
    newRay.direction = refracted;
  }
  rtTrace(topGroup, newRay, newPld);
  pld.color = newPld.color * glassParams.albedo;
}

// ====================== Disney =========================

rtDeclareVariable(DisneyParams, disneyParams, , );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtBuffer<LightParams> lights;

RT_PROGRAM void disney() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }

  // TODO: sample from light directly before bsdf
  // closest hit should be different for opaque and transparent

  // sample
  float3 N = faceforward(shadingNormal, -ray.direction, geoNormal);
  float3 V = -ray.direction;
  float3 L;
  float3 H;
  float diffuseRatio = 0.5f * (1.0f - disneyParams.metallic);
  Onb onb(N);
  if (rand(pld.randSeed) < diffuseRatio) { // diffuse
    cosine_sample_hemisphere(rand(pld.randSeed), rand(pld.randSeed), L);
    onb.inverse_transform(L);
    H = normalize(L + V);
  } else { // specular
    float a = max(0.001f, disneyParams.roughness);
    float phi = rand(pld.randSeed) * 2.0f * M_PIf;
    float random = rand(pld.randSeed);
    float cosTheta = sqrtf((1.f - random) / (1.0f + (a * a - 1.f) * random));
    float sinTheta = sqrtf(1.0f - (cosTheta * cosTheta));
    float sinPhi = sinf(phi);
    float cosPhi = cosf(phi);
    H = make_float3(sinTheta*cosPhi, sinTheta*sinPhi, cosTheta);
    onb.inverse_transform(H);
    L = normalize(2.0f * dot(V, H) * H - V);
  }
  Ray newRay(frontHitPoint, L, rayTypeRadiance, rayEpsilonT);
  Payload newPld;
  newPld.depth = pld.depth + 1;
  newPld.color = make_float3(1.f);
  newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
  rtTrace(topGroup, newRay, newPld);
  float3 lightColor = newPld.color;

  if (dot(N, L) <= 0.0f || dot(N, V) <= 0.0f) {
    pld.color = make_float3(0.f);
    return;
  }

  float pdf = disneyPdf(disneyParams, N, L, V, H);

  if (pdf < 0) {
    pld.color = make_float3(0.f);
    return;
  }

  float3 baseColor;
  if (disneyParams.albedoID == RT_TEXTURE_ID_NULL) {
    baseColor = disneyParams.color;
  } else {
    baseColor = make_float3(optix::rtTex2D<float4>(disneyParams.albedoID, texcoord.x, texcoord.y));
  }
  float3 brdf = disneyEval(disneyParams, baseColor, N, L, V, H, onb);

  pld.color = brdf * lightColor / pdf;
}

// ====================== light ==========================

rtDeclareVariable(LightParams, lightParams, , );

RT_PROGRAM void light() {
  pld.color = lightParams.emission;// * clamp(dot(ray.direction, shadingNormal), 0.f, 1.f);
}

// ====================== anyhit =========================

RT_PROGRAM void basicAnyHit() {
  pld.isShadow = true;
  rtTerminateRay();
}
