#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(PayloadRadiance, pldR, rtPayload, );
rtDeclareVariable(rtObject, topObject, , );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint, rayMaxDepth, , );
rtDeclareVariable(uint, rayTypeRadience, , );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(float, rayEpsilonT, , );
rtDeclareVariable(float, rayMinIntensity, , );
rtDeclareVariable(float3, absortColor, , );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );
rtDeclareVariable(float3, mtlColor, , );

// =================== lambertian ======================

rtDeclareVariable(float, nScatter, , );

RT_PROGRAM void lambertian() {
  if (pldR.depth > rayMaxDepth || length(pldR.color) < rayMinIntensity) {
    pldR.color = absortColor;
    return;
  }
  float3 P = ray.origin + t * ray.direction;
  int nNewRay = int(nScatter / pldR.depth + 1);
  float3 tmpColor = { 0.f, 0.f, 0.f };
  for (int i = 0; i < nNewRay; ++i) {
    float3 rayOrigin = P;
    float3 rayDirection = geoNormal + randInUnitSphere(pldR.randSeed);
    Ray newRay(rayOrigin, rayDirection, rayTypeRadience, rayEpsilonT);
    PayloadRadiance newPldR;
    newPldR.color = make_float3(1.f, 1.f, 1.f);
    newPldR.depth = pldR.depth + 1;
    newPldR.randSeed = pldR.randSeed + newPldR.depth * i;
    rtTrace(topObject, newRay, newPldR);
    tmpColor += newPldR.color;
  }
  tmpColor /= nNewRay;
  pldR.color *= tmpColor;
  pldR.color *= mtlColor;
}

// ====================== light ======================

rtDeclareVariable(float3, lightColor, , );

RT_PROGRAM void light() {
  pldR.color *= lightColor;
}
