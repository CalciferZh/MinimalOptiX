#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtDeclareVariable(rtObject, topGroup, , );
rtDeclareVariable(uint, rayMaxDepth, , );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(float, rayEpsilonT, , );
rtDeclareVariable(float, rayMinIntensity, , );
rtDeclareVariable(float3, absorbColor, , );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );
rtDeclareVariable(float3, shadingNormal,   attribute shadingNormal, );
rtDeclareVariable(float3, frontHitPoint, attribute frontHitPoint, );

// =================== lambertian ======================

rtDeclareVariable(LambertianParams, lambParams, , );

RT_PROGRAM void lambertian() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  int nNewRay = lambParams.nScatter;
  if (pld.depth > lambParams.scatterMaxDepth) {
    nNewRay = 1;
  }
  float3 tmpColor = { 0.f, 0.f, 0.f };
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.depth = pld.depth + 1;
  for (int i = 0; i < nNewRay; ++i) {
    newRay.direction = normalize(geoNormal + randInUnitSphere(pld.randSeed));
    newPld.color = make_float3(1.f);
    newPld.randSeed = tea<16>(pld.randSeed, newPld.depth * lambParams.nScatter + i);
    rtTrace(topGroup, newRay, newPld);
    tmpColor += newPld.color;
  }
  tmpColor /= nNewRay;
  pld.color *= tmpColor;
  pld.color *= lambParams.albedo;
}

// ====================== metal ==========================

rtDeclareVariable(MetalParams, metalParams, , );

RT_PROGRAM void metal() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.direction = normalize(reflect(ray.direction, geoNormal) + metalParams.fuzz * randInUnitSphere(pld.randSeed));
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.color = make_float3(1.f);
  newPld.depth = pld.depth + 1;
  newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
  rtTrace(topGroup, newRay, newPld);
  pld.color *= newPld.color;
  pld.color *= metalParams.albedo;
}

// ====================== glass ==========================

rtDeclareVariable(GlassParams, glassParams, , );

RT_PROGRAM void glass() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }

  float3 reflected = reflect(ray.direction, geoNormal);
  float3 outwardNormal;
  float realRefIdx;
  float cosine;
  if (dot(ray.direction, geoNormal) > 0) {
    outwardNormal = -geoNormal;
    realRefIdx = glassParams.refIdx;
    cosine = dot(ray.direction, geoNormal);
    cosine = sqrt(1 - glassParams.refIdx * glassParams.refIdx * (1 - cosine * cosine));
  } else {
    outwardNormal = geoNormal;
    realRefIdx = 1.f / glassParams.refIdx;
    cosine = dot(-ray.direction, geoNormal);
  }
  float3 refracted;
  float reflectProb;
  int nNewRay;
  if (refract(ray.direction, outwardNormal, realRefIdx, refracted)) {
    reflectProb = schlick(cosine, glassParams.refIdx);
    nNewRay = glassParams.nScatter;
  } else {
    reflectProb = 1.f;
    nNewRay = 1;
  }
  if (pld.depth > glassParams.scatterMaxDepth) {
    nNewRay = 1;
  }
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.depth = pld.depth + 1;
  float3 tmpColor = { 0.f, 0.f, 0.f };
  for (int i = 0; i < nNewRay; ++i) {
    if (rand(pld.randSeed) < reflectProb) {
      newRay.direction = reflected;
    } else {
      newRay.direction = refracted;
    }
    newPld.color = make_float3(1.f, 1.f, 1.f);
    newPld.randSeed = tea<16>(pld.randSeed, newPld.depth * glassParams.nScatter + i);
    rtTrace(topGroup, newRay, newPld);
    tmpColor += newPld.color;
  }
  tmpColor /= nNewRay;
  pld.color *= tmpColor;
  pld.color *= glassParams.albedo;
}

// ====================== Disney =========================

rtDeclareVariable(DisneyParams, disneyParams, , );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

RT_PROGRAM void disney() {
  float3 baseColor;
  if (disneyParams.albedoID == RT_TEXTURE_ID_NULL) {
    baseColor = disneyParams.color;
  } else {
		const float3 texColor = make_float3(optix::rtTex2D<float4>(disneyParams.albedoID, texcoord.x, texcoord.y));
		baseColor = make_float3(texColor.x, texColor.y, texColor.z);
  }
  // if (disneyParams.BrdfType == NORMAL) {
  if (true) {
    // sample
    float3 N = faceforward(shadingNormal, -ray.direction, geoNormal);
    float3 V = -ray.direction;
    float3 L;
    float diffuseRatio = 0.5f * (1.0f - disneyParams.metallic);
    float r1 = rand(pld.randSeed);
    float r2 = rand(pld.randSeed);
    optix::Onb onb(N);
    if (rand(pld.randSeed) < diffuseRatio) { // diffuse
      cosine_sample_hemisphere(r1, r2, L);
      onb.inverse_transform(L);
    } else { // spect
      float a = max(0.001f, disneyParams.roughness);
      float phi = r1 * 2.0f * M_PIf;
      float cosTheta = sqrtf((1.f - r2) / (1.0f + (a * a - 1.f) * r2));
      float sinTheta = sqrtf(1.0f - (cosTheta * cosTheta));
      float sinPhi = sinf(phi);
      float cosPhi = cosf(phi);
      float3 half = make_float3(sinTheta*cosPhi, sinTheta*sinPhi, cosTheta);
      onb.inverse_transform(half);
      L = 2.0f * dot(V, half) * half - V;
    }
    Ray newRay(frontHitPoint, normalize(L), rayTypeRadiance, rayEpsilonT);
    Payload newPld;
    newPld.depth = pld.depth + 1;
    newPld.color = make_float3(1.f);
    newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
    rtTrace(topGroup, newRay, newPld);
    pld.color *= newPld.color;

    // probability for this light
    float specularAlpha = max(0.001f, disneyParams.roughness);
    float clearcoatAlpha = lerp(0.1f, 0.001f, disneyParams.clearcoatGloss);
    float specularRatio = 1.f - diffuseRatio;
    float3 half = normalize(L + V);
    float cosTheta = abs(dot(half, N));
    float pdfGTR2 = GTR2(cosTheta, specularAlpha) * cosTheta;
    float pdfGTR1 = GTR1(cosTheta, clearcoatAlpha) * cosTheta;
    // calculate diffuse and specular pdfs and mix ratio
    float ratio = 1.0f / (1.0f + disneyParams.clearcoat);
    float pdfSpec = lerp(pdfGTR1, pdfGTR2, ratio) / (4.0 * abs(dot(L, half)));
    float pdfDiff = abs(dot(L, N))* (1.0f / M_PIf);
    float pdf = diffuseRatio * pdfDiff + specularRatio * pdfSpec;

    if (pdf < 0) {
      pld.color *= 0.f;
      return;
    }

    // evaluate color
    float NDotL = dot(N, L);
    float NDotV = dot(N, V);
    if (NDotL <= 0.0f || NDotV <= 0.0f) {
      pld.color = make_float3(0.f);
      return;
    }
    float3 H = normalize(L + V);
    float NDotH = dot(N, H);
    float LDotH = dot(L, H);
    float3 Cdlin = baseColor;
    float Cdlum = 0.3f * Cdlin.x + 0.6f * Cdlin.y + 0.1f * Cdlin.z;
    float3 Ctint = Cdlum > 0.0f ? Cdlin / Cdlum : make_float3(1.f);
    float3 Cspec0 = lerp(disneyParams.specular * 0.08f * lerp(make_float3(1.f), Ctint, disneyParams.specularTint), Cdlin, disneyParams.metallic);
    float3 Csheen = lerp(make_float3(1.f), Ctint, disneyParams.sheenTint);
    float FL = SchlickFresnel(NDotL);
    float FV = SchlickFresnel(NDotV);
    float Fd90 = 0.5f + 2.f * LDotH * LDotH * disneyParams.roughness;
    float Fd = lerp(1.f, Fd90, FL) * lerp(1.f, Fd90, FV);
    float Fss90 = LDotH * LDotH * disneyParams.roughness;
    float Fss = lerp(1.0f, Fss90, FL) * lerp(1.0f, Fss90, FV);
    float ss = 1.25f * (Fss * (1.0f / (NDotL + NDotV) - 0.5f) + 0.5f);
    float aspect = sqrt(1 - disneyParams.anisotropic * 0.9f);
    float ax = max(.001f, sqrt(disneyParams.roughness) / aspect);
    float ay = max(.001f, sqrt(disneyParams.roughness) * aspect);
    float3 X = { 1.f, 0.f, 0.f };
    float3 Y = { 0.f, 1.f, 1.f };
    float Ds = GTR2_Aniso(NDotH, dot(H, X), dot(H, Y), ax, ay);
    // float a = max(0.001f, disneyParams.roughness);
    // float Ds = GTR2(NDotH, a);
    float FH = SchlickFresnel(LDotH);
    float3 Fs = lerp(Cspec0, make_float3(1.0f), FH);
    float roughg = sqrt(disneyParams.roughness*0.5f + 0.5f);
    float Gs = smithG_GGX(NDotL, roughg) * smithG_GGX(NDotV, roughg);
    float3 Fsheen = FH * disneyParams.sheen * Csheen;
    float Dr = GTR1(NDotH, lerp(0.1f, 0.001f, disneyParams.clearcoatGloss));
    float Fr = lerp(0.04f, 1.0f, FH);
    float Gr = smithG_GGX(NDotL, 0.25f) * smithG_GGX(NDotV, 0.25f);
    float3 tmpColor = ((1.0f / M_PIf) * lerp(Fd, ss, disneyParams.subsurface) * Cdlin + Fsheen)
      * (1.0f - disneyParams.metallic)
      + Gs * Fs * Ds + 0.25f * disneyParams.clearcoat * Gr * Fr * Dr;
    float3 finalColor = tmpColor * clamp(dot(N, L), 0.0f, 1.0f);

    pld.color *= finalColor / pdf;
  } else {
    return;
  }
}

// ====================== light ==========================

rtDeclareVariable(LightParams, lightParams, , );
// NOTE: some of light's attributes need to be computed mannually

RT_PROGRAM void light() {
  pld.color *= lightParams.emission;
}
