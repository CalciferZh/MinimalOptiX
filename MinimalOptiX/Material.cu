#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "disney.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtDeclareVariable(rtObject, topGroup, , );
rtDeclareVariable(uint, rayMaxDepth, , );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(uint, rayTypeShadow, , );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(float, rayEpsilonT, , );
rtDeclareVariable(float, rayMinIntensity, , );
rtDeclareVariable(float3, absorbColor, , );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float3, frontHitPoint, attribute frontHitPoint, );
rtDeclareVariable(float3, backHitPoint, attribute backHitPoint, );

// =================== lambertian ======================

rtDeclareVariable(LambertianParams, lambParams, , );

RT_PROGRAM void lambertian() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  int nNewRay = lambParams.nScatter;
  if (pld.depth > lambParams.scatterMaxDepth) {
    nNewRay = 1;
  }
  float3 tmpColor = { 0.f, 0.f, 0.f };
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.depth = pld.depth + 1;
  for (int i = 0; i < nNewRay; ++i) {
    newRay.direction = normalize(geoNormal + randInUnitSphere(pld.randSeed));
    newPld.color = make_float3(1.f);
    newPld.randSeed = tea<16>(pld.randSeed, newPld.depth * lambParams.nScatter + i);
    rtTrace(topGroup, newRay, newPld);
    tmpColor += newPld.color;
  }
  tmpColor /= nNewRay;
  pld.color = tmpColor * lambParams.albedo;
}

// ====================== metal ==========================

rtDeclareVariable(MetalParams, metalParams, , );

RT_PROGRAM void metal() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.direction = normalize(reflect(ray.direction, geoNormal) + metalParams.fuzz * randInUnitSphere(pld.randSeed));
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.color = make_float3(1.f);
  newPld.depth = pld.depth + 1;
  newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
  rtTrace(topGroup, newRay, newPld);
  pld.color = metalParams.albedo * newPld.color;
}

// ====================== glass ==========================

rtDeclareVariable(GlassParams, glassParams, , );

RT_PROGRAM void glass() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }

  float3 normal = shadingNormal;
	float cosThetaI = -dot(ray.direction, normal);
	float refIdx;
	if (cosThetaI > 0.f) {
		refIdx = glassParams.refIdx;
	} else {
		refIdx = 1.f / glassParams.refIdx;
		cosThetaI = -cosThetaI;
		normal = -normal;
	}

	float3 refracted;
  float totalReflection = !refract(refracted, ray.direction, normal, refIdx);
	float cosThetaT = -dot(normal, refracted);
	float reflectProb =  totalReflection ? 1.f : fresnel(cosThetaI, cosThetaT, refIdx);
  Ray newRay;
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.depth = pld.depth + 1;
  newPld.color = make_float3(1.f, 1.f, 1.f);
  newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
  if (rand(pld.randSeed) < reflectProb) {
    newRay.origin = frontHitPoint;
    newRay.direction = reflect(ray.direction, normal);
  } else {
    newRay.origin = backHitPoint;
    newRay.direction = refracted;
  }
  rtTrace(topGroup, newRay, newPld);
  pld.color = newPld.color * glassParams.albedo;
}

// ====================== Disney =========================

rtDeclareVariable(DisneyParams, disneyParams, , );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtBuffer<LightParams> lights;

RT_PROGRAM void disney() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }

  float3 N, L, V, H;
  N = faceforward(shadingNormal, -ray.direction, geoNormal);
  V = -ray.direction;
  float3 baseColor;
  if (disneyParams.albedoID == RT_TEXTURE_ID_NULL) {
    baseColor = disneyParams.color;
  } else {
    baseColor = make_float3(optix::rtTex2D<float4>(disneyParams.albedoID, texcoord.x, texcoord.y));
  }

  if (disneyParams.brdfType == GLASS) {
    float3 normal = shadingNormal;
    float cosThetaI = -dot(ray.direction, normal);
    float refIdx;
    if (cosThetaI > 0.f) {
      refIdx = 1.45f;
    } else {
      refIdx = 1.f / 1.45f;
      cosThetaI = -cosThetaI;
      normal = -normal;
    }

    float3 refracted;
    float totalReflection = !refract(refracted, ray.direction, normal, refIdx);
    float cosThetaT = -dot(normal, refracted);
    float reflectProb =  totalReflection ? 1.f : fresnel(cosThetaI, cosThetaT, refIdx);
    Ray newRay;
    newRay.ray_type = rayTypeRadiance;
    newRay.tmin = rayEpsilonT;
    newRay.tmax = RT_DEFAULT_MAX;
    Payload newPld;
    newPld.depth = pld.depth + 1;
    newPld.color = make_float3(1.f, 1.f, 1.f);
    newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
    if (rand(pld.randSeed) < reflectProb) {
      newRay.origin = frontHitPoint;
      newRay.direction = reflect(ray.direction, normal);
    } else {
      newRay.origin = backHitPoint;
      newRay.direction = refracted;
    }
    rtTrace(topGroup, newRay, newPld);
    pld.color = newPld.color * baseColor;
    return;
  }

  // direct light sample
  float3 directLightColor = make_float3(0.f);
  for (int i = 0; i < lights.size(); ++i) {
    LightParams light = lights[i];
    float3 pointOnLight;
    float3 normalOnLight;
    if (light.shape == SPHERE) {
      pointOnLight = light.position + randInUnitSphere(pld.randSeed) * light.radius;
      normalOnLight = normalize(pointOnLight - light.position);
    } else {
      pointOnLight = light.position + light.u * rand(pld.randSeed) + light.v * rand(pld.randSeed);
      normalOnLight = normalize(light.normal);
    }
    L = pointOnLight - frontHitPoint;
    float lightDst = length(L);
    L = normalize(L);
    if (dot(L, N) > 0.f && dot(L, normalOnLight) < 0.f) {
      Ray newRay(frontHitPoint, L, rayTypeShadow, rayEpsilonT, lightDst - rayEpsilonT);
      Payload newPld;
      newPld.depth = pld.depth + 1;
      newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
      newPld.attenuation = make_float3(1.f);
      rtTrace(topGroup, newRay, newPld);
      if (length(newPld.attenuation)) {
        H = normalize(L + V);
        float lightPdf = lightDst * lightDst / light.area / dot(normalOnLight, -L);
        float objPdf = disneyPdf(disneyParams, N, L, V, H);
        if (lightPdf > 0 && objPdf > 0) {
          float3 brdf = disneyEval(disneyParams, baseColor, N, L, V, H);
          directLightColor += powerHeuristic(lightPdf, objPdf) * brdf * light.emission / max(0.001f, lightPdf);
        }
      }
    }
  }

  float3 indirectColor = make_float3(0.f);
  disneySample(pld.randSeed, disneyParams, N, L, V, H);
  if (dot(N, L) > 0.0f && dot(N, V) > 0.0f) {
    Ray newRay(frontHitPoint, L, rayTypeRadiance, rayEpsilonT);
    Payload newPld;
    newPld.depth = pld.depth + 1;
    newPld.color = make_float3(1.f);
    newPld.randSeed = tea<16>(pld.randSeed, newPld.depth);
    rtTrace(topGroup, newRay, newPld);

    float pdf = disneyPdf(disneyParams, N, L, V, H);
    if (pdf > 0) {
      float3 brdf = disneyEval(disneyParams, baseColor, N, L, V, H);
      indirectColor = brdf * newPld.color / pdf;
    }
  }

  pld.color = indirectColor + directLightColor;
}

RT_PROGRAM void disneyAnyHit() {
  if (disneyParams.brdfType != GLASS) {
    pld.attenuation = make_float3(0.f);
    rtTerminateRay();
  }
}

// ====================== light ==========================

rtDeclareVariable(LightParams, lightParams, , );

RT_PROGRAM void light() {
  pld.color = lightParams.emission;
}

