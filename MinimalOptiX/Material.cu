#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtDeclareVariable(rtObject, topObject, , );
rtDeclareVariable(uint, rayMaxDepth, , );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(float, rayEpsilonT, , );
rtDeclareVariable(float, rayMinIntensity, , );
rtDeclareVariable(float3, absorbColor, , );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );

// =================== lambertian ======================

rtDeclareVariable(LambertianParams, lambParams, , );

RT_PROGRAM void lambertian() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  int nNewRay = lambParams.nScatter / pld.depth + 1;
  if (pld.depth > lambParams.scatterMaxDepth) {
    nNewRay = 1;
  }
  float3 tmpColor = { 0.f, 0.f, 0.f };
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.depth = pld.depth + 1;
  for (int i = 0; i < nNewRay; ++i) {
    newRay.direction = geoNormal + randInUnitSphere(pld.randSeed);
    newPld.color = make_float3(1.f, 1.f, 1.f);
    newPld.randSeed = pld.randSeed + newPld.depth * i;
    rtTrace(topObject, newRay, newPld);
    tmpColor += newPld.color;
  }
  tmpColor /= nNewRay;
  pld.color *= tmpColor;
  pld.color *= lambParams.albedo;
}

// ====================== metal ==========================

rtDeclareVariable(MetalParams, metalParams, , );

RT_PROGRAM void metal() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  Ray newRay;
  newRay.origin = ray.origin + t * ray.direction;
  newRay.direction = reflect(ray.direction, geoNormal) + metalParams.fuzz * randInUnitSphere(pld.randSeed);
  newRay.ray_type = rayTypeRadiance;
  newRay.tmin = rayEpsilonT;
  newRay.tmax = RT_DEFAULT_MAX;
  Payload newPld;
  newPld.color = make_float3(1.f, 1.f, 1.f);
  newPld.depth = pld.depth + 1;
  newPld.randSeed = pld.randSeed + pld.depth;
  rtTrace(topObject, newRay, newPld);
  pld.color *= newPld.color;
  pld.color *= metalParams.albedo;
}

// ====================== light ==========================

rtDeclareVariable(float3, lightColor, , );

RT_PROGRAM void light() {
  pld.color *= lightColor;
}
