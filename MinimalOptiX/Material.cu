#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "structures.h"
#include "utils_device.h"

using namespace optix;

rtDeclareVariable(Payload, pld, rtPayload, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtDeclareVariable(rtObject, topObject, , );
rtDeclareVariable(uint, rayMaxDepth, , );
rtDeclareVariable(uint, rayTypeRadiance, , );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(float, rayEpsilonT, , );
rtDeclareVariable(float, rayMinIntensity, , );
rtDeclareVariable(float3, absorbColor, , );
rtDeclareVariable(float3, geoNormal, attribute geoNormal, );

// =================== lambertian ======================

rtDeclareVariable(LambertianParams, lambParams, , );

RT_PROGRAM void lambertian() {
  if (pld.depth > rayMaxDepth || length(pld.color) < rayMinIntensity) {
    pld.color = absorbColor;
    return;
  }
  float3 P = ray.origin + t * ray.direction;
  int nNewRay = lambParams.nScatter / pld.depth + 1;
  float3 tmpColor = { 0.f, 0.f, 0.f };
  for (int i = 0; i < nNewRay; ++i) {
    float3 rayOrigin = P;
    float3 rayDirection = geoNormal + randInUnitSphere(pld.randSeed);
    Ray newRay(rayOrigin, rayDirection, rayTypeRadiance, rayEpsilonT);
    Payload newPld;
    newPld.color = make_float3(1.f, 1.f, 1.f);
    newPld.depth = pld.depth + 1;
    newPld.randSeed = pld.randSeed + newPld.depth * i;
    rtTrace(topObject, newRay, newPld);
    tmpColor += newPld.color;
  }
  tmpColor /= nNewRay;
  pld.color *= tmpColor;
  pld.color *= lambParams.albedo;
}

// ====================== light ======================

rtDeclareVariable(float3, lightColor, , );

RT_PROGRAM void light() {
  pld.color *= lightColor;
}
